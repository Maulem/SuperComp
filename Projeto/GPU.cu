#include "hip/hip_runtime.h"
%%writefile GPU.cu

#include <iostream>
#include <algorithm>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>

using namespace std;
using namespace std::chrono;

struct subsequenceIndex {
    int start;
    int end;
};

std::vector<subsequenceIndex> generateSubsequences(int max_size) {
    vector<subsequenceIndex> indexes;
    subsequenceIndex index;

    for(int i = 0; i < max_size; i++) {
        for(int j = 1; j <= max_size; j++) {
            index.start = i;
            index.end = j;
            if (i < j)
                indexes.push_back(index);
        }
    }
    return indexes;
}


struct nucleobaseMatch{
    __host__ __device__
    int operator()(const char& dna_a, const char& dna_b) {
        if (dna_a != 'A' && dna_a != 'T' && dna_a != 'C' && dna_a != 'G' && dna_a != '-') {
            return 0;
        }
        if (dna_b != 'A' && dna_b != 'T' && dna_b != 'C' && dna_b != 'G' && dna_b != '-') {
            return 0;
        }
        if (dna_a == dna_b) {
            return 2;
        }
        return -1;
    }
};

int main() {

    // Declara as variaveis de contagem de tempo
    steady_clock::time_point init_time, final_time;

    // Inicia a contagem de tempo
    init_time = steady_clock::now();
    
    // Declara as variaveis usadas
    std::string dna_a, dna_b;
    int size_a, size_b;
    
    // Recebe e calcula o tamanho dos DNAs
    std::cin >> dna_a >> dna_b;
    size_a = dna_a.length();
    size_b = dna_b.length();

    // Gera as variaveis usadas pra checar as subsequencias
    int score = 0, max_score = 0;
    int min_size = min(size_a,size_b);
    int max_size = max(size_a,size_b);

    // Gera as subsequencias possiveis diferentes de 0
    std::vector<subsequenceIndex> smaller_index_list = generateSubsequences(min_size), bigger_index_list = generateSubsequences(max_size);

    // Cria os vetores que vao ser usados na GPU
    thrust::device_vector<char> smaller_gpu(min_size), bigger_gpu(max_size);

    // Salva a menor string no menor vetor da GPU
    for(int i = 0; i < min_size; i++) {
        if (size_a < size_b) {
            smaller_gpu[i] = dna_a[i];
        }
        else {
            smaller_gpu[i] = dna_b[i];
        }
        //cout << smaller_gpu[i];
    }
    //cout << endl;

    // Salva a maior string no maior vetor da GPU
    for(int i = 0; i < max_size; i++) {
        if (size_a > size_b) {
            bigger_gpu[i] = dna_a[i];
        }
        else {
            bigger_gpu[i] = dna_b[i];
        }
        //cout << bigger_gpu[i];
    }
    //cout << endl;

    // Varre todos os indices
    for (auto& small_index : smaller_index_list) {
        for(auto& big_index : bigger_index_list) {
            // Se os indices nao tem o mesmo tamanho descarta fora 
            if (small_index.end - small_index.start == big_index.end - big_index.start) {
                // Cria o vetor que vai receber a pontuacao da comparacao de duas sequencias com o menor tamanho do menor indice
                thrust::device_vector<char> sequence_score(small_index.end);

                // Compara 2 vetores usando uma operacao e salva no vetor final os resultados
                thrust::transform(
                    smaller_gpu.begin() + small_index.start,    // Inicio vetor 1
                    smaller_gpu.end() + small_index.end,        // Fim vetor 1
                    bigger_gpu.begin() + big_index.start,       // Inicio vetor 2
                    sequence_score.begin(),                     // Inicio vetor final
                    nucleobaseMatch()                           // Operacao a ser feita
                );
                
                // Calcula o tamanho da menor subsequencia
                int dif = small_index.end - small_index.start;

                // Soma os valores do vetor comecando no inicio ate o tamanho da menor subsequencia
                score = thrust::reduce(sequence_score.begin(), sequence_score.begin() + dif, score, thrust::plus<int>());

                // Atualiza a novo pontuacao maxima
                if (max_score < score) {
                    // cout << "----------------------------------------" << endl;
                    // cout << small_index.start << " " << small_index.end << endl;
                    // for(int i = small_index.start; i < small_index.end; i++) {
                    //     cout << smaller_gpu[i];
                    // }
                    // cout << endl;
                    // cout << big_index.start << " " << big_index.end << endl;
                    // for(int i = big_index.start; i < big_index.end; i++) {
                    //     cout << bigger_gpu[i];
                    // }
                    // cout << endl;
                    // for (int i = 0; i < dif; i++) {
                    //     cout << (int)sequence_score[i];
                    // }
                    // cout << endl;
                    // cout << score << endl;
                    // cout << "----------------------------------------" << endl;
                    max_score = score;
                }
                // Zera o score por precaucao
                score = 0;
            }
        }
    }
    
    // Para a contagem de tempo
    final_time = steady_clock::now();

    // Printa os tamanhos das sequencias
    std::cout << size_a << std::endl;
    std::cout << size_b << std::endl;

    // Printa a duracao
    auto ms = duration_cast<milliseconds>(final_time - init_time);
    std::cout << (double)ms.count() / 1000 << std::endl;

    //std::cout << "Score: " << max_score << endl;

    return 0;
}